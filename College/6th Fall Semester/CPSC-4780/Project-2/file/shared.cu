#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <numeric>

typedef float real_t;

const size_t THREAD_PER_BLOCK = 512;

void random_ints(real_t *a, int size) {
    for (int i = 0; i < size; i++) {
        a[i] = drand48();
    }
}

__global__ void partial_dot(const real_t *v1, const real_t *v2, real_t *out, int N) {
    __shared__ real_t cache[THREAD_PER_BLOCK];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[threadIdx.x] = 0.f;
    while (i < N) {
        cache[threadIdx.x] += v1[i] * v2[i];
        i += gridDim.x * blockDim.x;
    }
    __syncthreads();
    // required because later on the current thread is accessing
    // data written by another thread
    i = THREAD_PER_BLOCK / 2;
    while (i > 0) {
        if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }

    if (threadIdx.x == 0) out[blockIdx.x] = cache[0];
}

//------------------------------------------------------------------------------
int main(int argc, char **argv) {

    const size_t ARRAY_SIZE = 16777216;
    const int BLOCKS = 64;
    const int THREADS_PER_BLOCK = THREAD_PER_BLOCK;
    const size_t SIZE = ARRAY_SIZE * sizeof(real_t);

    // device storage
    //real_t *dev_v1 = 0; // vector 1
    //real_t *dev_v2 = 0; // vector 2
  //  real_t *dev_vout = 0; // partial redution = number of blocks

    hipMalloc((void **) &dev_v1, SIZE);
    hipMalloc((void **) &dev_v2, SIZE);
    hipMalloc((void **) &dev_vout, BLOCKS * sizeof(real_t));
    dev_v1 = (real_t *) malloc(size);
    dev_v2 = (real_t *) malloc(size);
    dev_vout = (real_t *) malloc(sizeof(real_t));

    // host storage
    std::vector <real_t> host_v1(ARRAY_SIZE);
    std::vector <real_t> host_v2(ARRAY_SIZE);
    std::vector <real_t> host_vout(BLOCKS);

    random_ints(dev_v1, ARRAY_SIZE);
    random_ints(dev_v2, ARRAY_SIZE);

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventRecord(start1, 0);
    float cputime;
    // execute kernel
    partial_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_v1, dev_v2, dev_vout, ARRAY_SIZE);

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy(&host_vout[0], dev_vout, BLOCKS * sizeof(real_t), hipMemcpyDeviceToHost);

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: " << std::accumulate(host_vout.begin(), host_vout.end(), real_t(0)) << std::endl;

    hipEventCreate(&stop1);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&cputime, start1, stop1);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    // free memory
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_vout);

    return 0;
}
