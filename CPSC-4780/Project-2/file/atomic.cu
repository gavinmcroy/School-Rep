
#include <hip/hip_runtime.h>
#include <iostream>

#define N 16777216
#define THREADS_PER_BLOCK 512

typedef float real_t;

void random_ints(real_t *a, int size) {
    for (int i = 0; i < size; i++) {
        a[i] = drand48();
    }
}

__global__ void dot(real_t *a, real_t *b, real_t *c) {
    __shared__ real_t temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];
    __syncthreads();
    if (0 == threadIdx.x) {
        real_t sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
            sum += temp[i];
        atomicAdd(c, sum);
    }
}

int main(void) {
    real_t *a, *b, *c; // host copies of a, b, c
    real_t *dev_a, *dev_b, *dev_c; // device copies of a, b, c
    int size = N * sizeof(real_t); // we need space for N ints
// allocate device copies of a, b, c
    hipMalloc((void **) &dev_a, size);
    hipMalloc((void **) &dev_b, size);
    hipMalloc((void **) &dev_c, sizeof(real_t));
    a = (real_t *) malloc(size);
    b = (real_t *) malloc(size);
    c = (real_t *) malloc(sizeof(real_t));
    random_ints(a, N);
    random_ints(b, N);

    // copy inputs to device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    // launch dot() kernel
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventRecord(start1, 0);
    float cputime;
    dot<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, dev_c);

    // copy device result back to host copy of c
    hipMemcpy(c, dev_c, sizeof(real_t), hipMemcpyDeviceToHost);
    std::cout << *c << std::endl;

    hipEventCreate(&stop1);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&cputime, start1, stop1);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
